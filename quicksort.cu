#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"
#include <pthread.h>
__global__ void split(int N, int* input, int* left, int* right, int* leftcount, int* rightcount);


struct node{
	int* array;
	int numElements;
	node* left;
	node* right;
};

struct node2{
	int* array;
	int numElements;
	int before;
};

void* recursive_helper(void* current_node);
void* recursive_helper2(void* current_node);
void quick_sort(int N, int* input);
void quick_sort2(int N, int* input);

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];

	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}

	//YOUR CODE HERE
	quick_sort2(size,A);


	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}

	free(A);

	return 0;
}


__global__
void split(int N, int* input, int* left, int* right, int* leftcount, int* rightcount){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  //*leftcount = 0;
  //*rightcount = 0;
  __syncthreads();
  int splitter = input[0];
  for (int i = index+1; i < N; i+= stride){
    if(input[i] > splitter){
      right[atomicAdd(rightcount,1)] = input[i];
    }else{
      left[atomicAdd(leftcount,1)] = input[i];
    }
  }
    //*leftcount = local_left_count;
    //		*rightcount = local_right_count;

}


int* output;
int output_count;
int stream_cnt;
hipStream_t streams [50000];
	
void quick_sort2(int N, int* input){
	node2* rootptr;
	
	hipMallocManaged(&rootptr,sizeof(node2));
	node2 root = *rootptr;
	hipMallocManaged(&rootptr->array, sizeof(int)*N);
	hipMemcpy(rootptr->array,input,sizeof(int)*N,hipMemcpyHostToDevice);
	rootptr->numElements = N;
	hipMallocManaged(&output,sizeof(int)*N);
	recursive_helper2(&rootptr);
	hipMemcpy(input,output,N*sizeof(int),hipMemcpyDeviceToHost);
	
	hipFree(output);
}

void* recursive_helper2(void* ptr){
	
	node2* current_node = *((node2**)ptr);
			
	if(((node2*)current_node)->numElements == 1){
		output[((node2*)current_node)->before] = ((node2*)current_node)->array[0];
		hipFree(((node2*)current_node)->array);
		hipFree(current_node);
		return NULL;
	}if(((node2*)current_node)->numElements == 0){
		hipFree(((node2*)current_node)->array);
		hipFree(current_node);
		return NULL;
	}
	node2* left;
	node2* right;

	while(hipMallocManaged(&left, sizeof(node2)));
	while(hipMallocManaged(&left->array, sizeof(int)*(((node2*)current_node)->numElements)));
	while(hipMallocManaged(&right, sizeof(node2)));
	while(hipMallocManaged(&right->array, sizeof(int)*(((node2*)current_node)->numElements)));

	int numBlocks = (((node2*)current_node)->numElements + 256 - 1) /256;
	
	int streamID = __atomic_fetch_add(&stream_cnt, 1, __ATOMIC_SEQ_CST);
	hipStreamCreate(&streams[streamID]);
	
	split<<<1, 256, 0, streams[streamID]>>>(((node2*)current_node)->numElements, ((node2*)current_node)->array, left->array, right->array, &left->numElements, &right->numElements);
	hipStreamSynchronize(streams[streamID]);
	
	left->array[left->numElements] = ((node2*)current_node)->array[0];
	left->numElements ++;
	
	left->before = ((node2*)current_node)->before;
	right->before = ((node2*)current_node)->before + left->numElements;

	pthread_t left_thread, right_thread;

	hipDeviceSynchronize();
	hipFree(((node2*)current_node)->array);
	hipFree(((node2*)current_node));


	pthread_create(&left_thread, NULL, recursive_helper2, (void*) &left);
//	pthread_join(left_thread,NULL);
	pthread_create(&right_thread, NULL, recursive_helper2, (void*) &right);

	pthread_join(left_thread,NULL);
	pthread_join(right_thread,NULL);
	

	return NULL;

}
