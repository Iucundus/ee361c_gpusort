#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"

__global__ void split(int N, int* input, int* left, int* right, int* leftcount, int* rightcount);

struct node{
	int* array;
	int numElements;
	node* left;
	node* right;
};
void recursive_helper(node* current_node);
int* quick_sort(int N, int* input);

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];

	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}

	//YOUR CODE HERE
	A = quick_sort(size,A);


	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}

	free(A);

	return 0;
}




__global__
void split(int N, int* input, int* left, int* right, int* leftcount, int* rightcount){
  int index = threadIdx.x;
  int stride = blockDim.x;

  __shared__ int local_left_count;
  __shared__ int local_right_count;
  local_left_count = 0;
  local_right_count = 0;
  __syncthreads();
  int splitter = input[0];

  for (int i = index+1; i < N; i+= stride){
    if(input[i] > splitter){
      right[atomicAdd(&local_right_count,1)] = input[i];
    }else{
      left[atomicAdd(&local_left_count,1)] = input[i];
    }
  }
  __syncthreads();
  if(index == 0){
    left[atomicAdd(&local_left_count,1)] = input[0];
		*leftcount = local_left_count;
		*rightcount = local_right_count;
  }

}


int* output;
int output_count;
int* quick_sort(int N, int* input){
	node root;
	root.array = input;
	root.numElements = N;
	output = (int*)malloc(sizeof(int)*N);
	output_count = 0;
	recursive_helper(&root);
	return output;
}

void recursive_helper(node* the_node){
	node current_node = *the_node;
	printf("%d ",current_node.numElements);
	if(current_node.numElements < 2){
		output[output_count] = current_node.array[0];
		output_count++;
		return;
	}
	current_node.left = (node*)malloc(sizeof(node));
	current_node.right = (node*)malloc(sizeof(node));
	node left_node = *current_node.left;
	node right_node = *current_node.right;
	hipMallocManaged(&left_node.array, sizeof(int)*current_node.numElements);
	hipMallocManaged(&right_node.array, sizeof(int)*current_node.numElements);
	//TODO add break condition
	

	split<<<1,1>>>(current_node.numElements, current_node.array, left_node.array, right_node.array, &left_node.numElements, &right_node.numElements);
	hipDeviceSynchronize();
	printf("%d left",left_node.numElements);
	printf("%d right",right_node.numElements);
	recursive_helper(current_node.left);
	recursive_helper(current_node.right);

}
