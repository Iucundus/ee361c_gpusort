#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"

__global__ void split(int N, int* input, int* left, int* right, int* leftcount, int* rightcount);

struct node{
	int* array;
	int numElements;
	node* left;
	node* right;
};
void recursive_helper(node* current_node);
void quick_sort(int N, int* input);

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];

	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}

	//YOUR CODE HERE
	quick_sort(size,A);


	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}

	free(A);

	return 0;
}




__global__
void split(int N, int* input, int* left, int* right, int* leftcount, int* rightcount){
  int index = threadIdx.x;
  int stride = blockDim.x;
  __shared__ int local_left_count;
  __shared__ int local_right_count;
  local_left_count = 0;
  local_right_count = 0;
  __syncthreads();
  int splitter = input[0];
  for (int i = index+1; i < N; i+= stride){
    if(input[i] > splitter){
      right[atomicAdd(&local_right_count,1)] = input[i];
    }else{
      left[atomicAdd(&local_left_count,1)] = input[i];
    }
  }
  __syncthreads();
    if(index == 0) left[atomicAdd(&local_left_count,1)] = input[0];
  __syncthreads();
    *leftcount = local_left_count;
		*rightcount = local_right_count;

}


int* output;
int output_count;
void quick_sort(int N, int* input){

	node* rootptr;
	hipMallocManaged(&rootptr, sizeof(node));
	node root = *rootptr;
	hipMallocManaged(&root.array, sizeof(int)*N);
	hipMemcpy(root.array,input,sizeof(int)*N,hipMemcpyHostToDevice);
	root.numElements = N;
	hipMallocManaged(&output,sizeof(int)*N);
	output_count = 0;
	recursive_helper(&root);
	hipMemcpy(input,output,N*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(output);
}

void recursive_helper(node* current_node){
	//printf("%d total ",current_node->numElements);

	//printf("%d splitter ",current_node->array[0]);
	if(current_node->numElements == 1){
		output[output_count] = current_node->array[0];
		output_count++;
		hipFree(current_node->array);
		hipFree(current_node);
		return;
	}if(current_node->numElements == 0){
	       hipFree(current_node->array);
       	hipFree(current_node);
		return;
	}
	hipMallocManaged(&current_node->left,sizeof(node));
	hipMallocManaged(&current_node->right,sizeof(node));

	hipMallocManaged(&current_node->left->array, sizeof(int)*current_node->numElements);
	hipMallocManaged(&current_node->right->array, sizeof(int)*current_node->numElements);
	//printf("%d", current_node->left->array[0]);


	split<<<1,256>>>(current_node->numElements, current_node->array, current_node->left->array, current_node->right->array, &current_node->left->numElements, &current_node->right->numElements);
	hipDeviceSynchronize();
	recursive_helper(current_node->left);
	recursive_helper(current_node->right);
	hipFree(current_node->array);
	hipFree(current_node);

}
