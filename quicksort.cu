#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"
#include <pthread.h>
__global__ void split(int N, int* input, int* left, int* right, int* leftcount, int* rightcount);


struct node{
	int* array;
	int numElements;
	node* left;
	node* right;
};

struct node2{
	int* array;
	int numElements;
	int before;
};

void* recursive_helper(void* current_node);
void* recursive_helper2(void* current_node);
void quick_sort(int N, int* input);
void quick_sort2(int N, int* input);

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];

	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}

	//YOUR CODE HERE
	quick_sort2(size,A);


	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}

	free(A);

	return 0;
}


__global__
void split(int N, int* input, int* left, int* right, int* leftcount, int* rightcount){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  //*leftcount = 0;
  //*rightcount = 0;
  __syncthreads();
  int splitter = input[0];
  for (int i = index+1; i < N; i+= stride){
    if(input[i] > splitter){
      right[atomicAdd(rightcount,1)] = input[i];
    }else{
      left[atomicAdd(leftcount,1)] = input[i];
    }
  }
    //*leftcount = local_left_count;
    //		*rightcount = local_right_count;

}


int* output;
int output_count;
int stream_cnt;
void quick_sort(int N, int* input){

	node* rootptr;
	hipMallocManaged(&rootptr, sizeof(node));
	node root = *rootptr;
	hipMallocManaged(&root.array, sizeof(int)*N);
	hipMemcpy(root.array,input,sizeof(int)*N,hipMemcpyHostToDevice);
	root.numElements = N;
	hipMallocManaged(&output,sizeof(int)*N);
	output_count = 0;
	
	pthread_t root_thread;
	void* temp_ptr = (void*)  &root;

	pthread_create(&root_thread, NULL, recursive_helper, (void*)temp_ptr );
	//pthread_create(&root_thread, NULL, recursive_helper
	
	//recursive_helper(&root);
	pthread_join(root_thread, NULL);
	hipFree(rootptr);	
	hipMemcpy(input,output,N*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(output);
}
hipStream_t streams [50000];
void *recursive_helper(void* ptr){
	
	node* current_node = (node*) ptr;
	printf("%d total\n ", current_node->numElements);
	
	if( current_node->numElements == 1){
		output[output_count] = current_node->array[0];
		output_count++;
		//hipFree(current_node->array);
		//hipFree(current_node);
		printf("exiting insert\n");
		return NULL;
	}if( current_node->numElements == 0){
		//hipFree(current_node->array);
       		//hipFree(current_node);
		printf("exiting 0\n");
		return NULL;
	}
	
	while( 0 != hipMallocManaged( &current_node->left, sizeof(node)));
	while( 0 != hipMallocManaged( &current_node->right, sizeof(node)));
	
	while( 0 != hipMallocManaged(&current_node->left->array, sizeof(int)*current_node->numElements));
	while( 0 != hipMallocManaged(&current_node->right->array, sizeof(int)*current_node->numElements));
	
	
	current_node->left->numElements = 0;
	current_node->right->numElements = 0;
	int numBlocks = (current_node->numElements + 256 -1) / 256;
	printf("calling split\n");
	
	
	int streamID = __atomic_fetch_add(&stream_cnt, 1, __ATOMIC_SEQ_CST);
	hipStreamCreate(&streams[streamID]);

	split<<<numBlocks,256, 0,streams[streamID]>>>(current_node->numElements, current_node->array, current_node->left->array, current_node->right->array, &current_node->left->numElements, &current_node->right->numElements);
	
	hipStreamSynchronize(streams[streamID]);
	printf("done splitting\n");
	printf("%d right Elem\n", current_node->right->numElements);
	printf("%d left Elem\n", current_node->left->numElements);

	pthread_t left_thread, right_thread;

	node* left = current_node->left;
	node* right = current_node->right;
	
	pthread_create(&left_thread, NULL, recursive_helper, (void*) left);
	pthread_join(left_thread,NULL);
	pthread_create(&right_thread, NULL, recursive_helper, (void*) right);
	
	//recursive_helper( (void*) current_node->left);
	//recursive_helper( (void*) current_node->right);
	pthread_join(left_thread, NULL);
	pthread_join(right_thread, NULL);
	
	hipFree(current_node->array);
	hipFree(current_node);
	printf("thread exiting");
	return NULL;
}
int done = 0;
int done_count;
void quick_sort2(int N, int* input){
	node2* rootptr;
	
	hipMallocManaged(&rootptr,sizeof(node2));
	node2 root = *rootptr;
	hipMallocManaged(&root.array, sizeof(int)*N);
	hipMemcpy(root.array,input,sizeof(int)*N,hipMemcpyHostToDevice);
	root.numElements = N;
	hipMallocManaged(&output,sizeof(int)*N);
	done_count = N;
	recursive_helper2(&root);
	while(!done);
	hipMemcpy(input,output,N*sizeof(int),hipMemcpyDeviceToHost);
	
	hipFree(output);
}

void* recursive_helper2(void* ptr){
	node2* current_node = (node2*) ptr;
	if(current_node->numElements == 1){
		output[current_node->before] = current_node->array[0];
		hipFree(current_node->array);
		hipFree(current_node);
		//if(current_node->before == done_count-2) done = 1;
		return NULL;
	}if(current_node->numElements == 0){
		hipFree(current_node->array);
		hipFree(current_node);
		return NULL;
	}
//	printf("%d\n", current_node->numElements);	
	node2* left;
	node2* right;

	while(hipMallocManaged(&left, sizeof(node2)));
	while(hipMallocManaged(&left->array, sizeof(int)*current_node->numElements));
	while(hipMallocManaged(&right, sizeof(node2)));
	while(hipMallocManaged(&right->array, sizeof(int)*current_node->numElements));

	int numBlocks = (current_node->numElements + 256 - 1) /256;
	
	int streamID = __atomic_fetch_add(&stream_cnt, 1, __ATOMIC_SEQ_CST);
	hipStreamCreate(&streams[streamID]);
	
	split<<<1, 256, 0, streams[streamID]>>>(current_node->numElements, current_node->array, left->array, right->array, &left->numElements, &right->numElements);
	hipStreamSynchronize(streams[streamID]);
	
	left->array[left->numElements] = current_node->array[0];
	left->numElements ++;

	left->before = current_node->before;
	right->before = current_node->before + left->numElements;

	pthread_t left_thread, right_thread;


	hipFree(current_node->array);
	hipFree(current_node);


	pthread_create(&left_thread, NULL, recursive_helper2, (void*) left);
//	pthread_join(left_thread,NULL);	
	pthread_create(&right_thread, NULL, recursive_helper2, (void*) right);

	pthread_join(left_thread,NULL);
	pthread_join(right_thread,NULL);
	
	//hipFree(current_node->array);
	//hipFree(current_node);
	
//	pthread_join(left_thread,NULL);
//	pthread_join(right_thread,NULL);

	return NULL;

}
