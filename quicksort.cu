#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"

__global__ void split(int N, int* input, int* left, int* right);

struct node{
	int* array;
	int numElements;
	node* left;
	node* right;
};

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];

	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}

	//YOUR CODE HERE
	int* input;
	int* left;
	int* right;

	hipMallocManaged(&left, size*sizeof(int));
	hipMallocManaged(&right, size*sizeof(int));
	hipMallocManaged(&input, size*sizeof(int));



	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}

	free(A);

	return 0;
}




__global__
void split(int N, int* input, int* left, int* right, int* leftcount int* rightcount){
  int index = threadIdx.x;
  int stride = blockDim.x;

  __shared__ int local_left_count;
  __shared__ int local_right_count;

  int splitter = input[0];

  for (int i = index+1; i < N; i+= stride){
    if(input[i] > splitter){
      right[atomicAdd(&rightcount,1)] = input[i];
    }else{
      left[atomicAdd(&leftcount,1)] = input[i];
    }
  }
  __syncthreads();
  if(index == 0){
    left[atomicAdd(&leftcount,1)] = input[0];
		*leftcount = local_left_count;
		*rightcount = local_right_count;
  }

}


int* output;
int output_count;
int* quick_sort(int N, int* input){
	node root;
	root.array = input;
	root.numElements = N;
	output = malloc(sizeof(int)*N);
	output_count = 0;
	recursive_helper(&root);
}

void recursive_helper(node* current_node){

	if(current_node.numElements < 2){
		output[output_count] = current_node.array[0];
		output_count++;
		return;
	}
	current_node.left = malloc(sizeof(node));
	current_node.right = malloc(sizeof(node));

	hipMallocManaged(&current_node.left.array, sizeof(int)*current_node.numElements);
	hipMallocManaged(&current_node.right.array, sizeof(int)*current_node.numElements);
	//TODO add break condition
	

	split<<<1,1>>>(size, current_node.array, current_node.left.array, current_node.right.array, &current_node.left.numElements, &current_node.right.numElements);

	recursive_helper(current_node.left);
	recursive_helper(current_node.right);

}
