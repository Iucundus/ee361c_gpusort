#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"
#include <pthread.h>
__global__ void split(int N, int* input, int* left, int* right, int* leftcount, int* rightcount);


struct node{
	int* array;
	int numElements;
	node* left;
	node* right;
};
void* recursive_helper(void* current_node);
void quick_sort(int N, int* input);

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];

	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}

	//YOUR CODE HERE
	quick_sort(size,A);


	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}

	free(A);

	return 0;
}




__global__
void split(int N, int* input, int* left, int* right, int* leftcount, int* rightcount){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  //*leftcount = 0;
  //*rightcount = 0;
  __syncthreads();
  int splitter = input[0];
  for (int i = index+1; i < N; i+= stride){
    if(input[i] > splitter){
      right[atomicAdd(rightcount,1)] = input[i];
    }else{
      left[atomicAdd(leftcount,1)] = input[i];
    }
  }
    //*leftcount = local_left_count;
    //		*rightcount = local_right_count;

}


int* output;
int output_count;
int stream_cnt;
void quick_sort(int N, int* input){

	node* rootptr;
	hipMallocManaged(&rootptr, sizeof(node));
	node root = *rootptr;
	hipMallocManaged(&root.array, sizeof(int)*N);
	hipMemcpy(root.array,input,sizeof(int)*N,hipMemcpyHostToDevice);
	root.numElements = N;
	hipMallocManaged(&output,sizeof(int)*N);
	output_count = 0;
	
	pthread_t root_thread;
	void* temp_ptr = (void*)  &root;

	pthread_create(&root_thread, NULL, recursive_helper, (void*)temp_ptr );
	//pthread_create(&root_thread, NULL, recursive_helper
	
	//recursive_helper(&root);
	pthread_join(root_thread, NULL);
	hipFree(rootptr);	
	hipMemcpy(input,output,N*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(output);
}
hipStream_t streams [5000];
void *recursive_helper(void* ptr){
	
	node* current_node = (node*) ptr;
	printf("%d total\n ", current_node->numElements);
	
	if( current_node->numElements == 1){
		output[output_count] = current_node->array[0];
		output_count++;
		//hipFree(current_node->array);
		//hipFree(current_node);
		printf("exiting insert\n");
		return NULL;
	}if( current_node->numElements == 0){
		//hipFree(current_node->array);
       		//hipFree(current_node);
		printf("exiting 0\n");
		return NULL;
	}
	
	while( 0 != hipMallocManaged( &current_node->left, sizeof(node)));
	while( 0 != hipMallocManaged( &current_node->right, sizeof(node)));
	
	while( 0 != hipMallocManaged(&current_node->left->array, sizeof(int)*current_node->numElements));
	while( 0 != hipMallocManaged(&current_node->right->array, sizeof(int)*current_node->numElements));
	
	
	current_node->left->numElements = 0;
	current_node->right->numElements = 0;
	int numBlocks = (current_node->numElements + 256 -1) / 256;
	printf("calling split\n");
	
	
	int streamID = __atomic_fetch_add(&stream_cnt, 1, __ATOMIC_SEQ_CST);
	hipStreamCreate(&streams[streamID]);

	split<<<numBlocks,256, 0,streams[streamID]>>>(current_node->numElements, current_node->array, current_node->left->array, current_node->right->array, &current_node->left->numElements, &current_node->right->numElements);
	
	hipStreamSynchronize(streams[streamID]);
	printf("done splitting\n");
	printf("%d right Elem\n", current_node->right->numElements);
	printf("%d left Elem\n", current_node->left->numElements);

	current_node->left->array[current_node->left->numElements] = current_node->array[0];
	current_node->left->numElements ++;

	pthread_t left_thread, right_thread;

	node* left = current_node->left;
	node* right = current_node->right;
	
	pthread_create(&left_thread, NULL, recursive_helper, (void*) left);
	//pthread_join(left_thread,NULL);
	pthread_create(&right_thread, NULL, recursive_helper, (void*) right);
	
	//recursive_helper( (void*) current_node->left);
	//recursive_helper( (void*) current_node->right);
	pthread_join(left_thread, NULL);
	pthread_join(right_thread, NULL);
	
	hipFree(current_node->array);
	hipFree(current_node);
	printf("thread exiting");
	return NULL;
}
