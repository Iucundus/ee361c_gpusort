#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"

__global__ void brickSort(int* array, int arrayLen, int p) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= arrayLen - 1)
		return;
	if ((p % 2 == 0) && (idx % 2 == 1))
		return;
	if ((p % 2 == 1) && (idx % 2 == 0))
		return;
	if (array[idx] > array[idx + 1]) {
		int tmp = array[idx + 1];
		array[idx + 1] = array[idx];
		array[idx] = tmp;
	}
}

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];
	
	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}
	
	//YOUR CODE HERE
	int* d_array;
	hipMalloc(&d_array, size * sizeof(int));
	hipMemcpy(d_array, A, size * sizeof(int), hipMemcpyHostToDevice);

	// To get the first element, first pass number is 0
	// Last pass number is probably (size - 2)
	// By zero-indexing, that means (size - 1) passes
	// If the largest element is in array[0], that gives (size - 1) passes to move it (size - 1) places to array[size - 1]
	for (int i = 0; i < size - 1; i++) {
		brickSort<<<(size + 1023)/1024, 1024>>>(d_array, size, i);
		hipDeviceSynchronize();
	}

	hipMemcpy(A, d_array, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_array);
	
	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}
	
	free(A);
	
	return 0;
}
