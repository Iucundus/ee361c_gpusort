#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];

	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}

	//YOUR CODE HERE


	int* x;
  int* buck0;
  int* buck1;
  int* out;
  int* prefix1;
  int* prefix2;

  hipMallocManaged(&x, size*sizeof(int));
  hipMallocManaged(&out, size*sizeof(int));
  hipMallocManaged(&buck0, size*sizeof(int));
  hipMallocManaged(&buck1, size*sizeof(int));
  hipMallocManaged(&prefix1, size*sizeof(int));
  hipMallocManaged(&prefix2, size*sizeof(int));

  hipMemcpy(x,A,size*sizeof(int), hipMemcpyHostToDevice);

  radix_sort<<<1,256>>>(size, x, out, buck0, buck1, prefix1, prefix2);

  hipDeviceSynchronize();

  hipMemcpy(A,out,size*sizeof(int),hipMemcpyDeviceToHost);

  hipFree(x);
  hipFree(buck0);
  hipFree(buck1);
  hipFree(prefix1);
  hipFree(prefix2);
  hipFree(out);




	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}

	free(A);

	return 0;
}






__global__
void radix_sort(int N, int* x, int* out, int* buck0, int* buck1, int* prefix1, int* prefix2){


    int index = threadIdx.x;
    int stride = blockDim.x;

    int mask = 1;
    int even = 1;

    int* temp;
    int* p1;
    int* p2;
    p1 = prefix1;
    p2 = prefix2;

    __shared__ int buck0count;
    __shared__ int buck1count;

    buck0count = 0;
    buck1count = 0;

    for(int count = 0; count < 32; count++){

      for(int i = index; i < N; i += stride){
        if(x[i] & mask){
          p1[i] = 1;
        }else{
          p1[i] = 0;
        }
      }

      __syncthreads();

      for (int offset = 1; offset < N; offset *= 2){

        for(int i = index; i < N; i += stride){
          if(i>=offset){

	    p2[i] = p1[i] + p1[i-offset];
	  }else{
	    p2[i] = p1[i];
	  }
        }
        temp = p1;
        p1 = p2;
        p2 = temp;

        __syncthreads();
      }

      buck0count = 0;
      buck1count = 0;
      __syncthreads();

      for(int i = index; i < N; i += stride){
        if(x[i] & mask){
          buck1[p1[i]-1] = x[i];
          atomicAdd(&buck1count,1);
        }else{
          buck0[i-p1[i]] = x[i];
          atomicAdd(&buck0count,1);
        }
      }
      __syncthreads();
      int i = 0;
      for(i = index; i < buck0count; i+= stride){
        x[i] = buck0[i];
      }
      __syncthreads();
      for(int j = index; j < buck1count; j+= stride){
        x[j+buck0count] = buck1[j];
      }

      ////////////

      mask <<= 1;
      __syncthreads();
    }

}
