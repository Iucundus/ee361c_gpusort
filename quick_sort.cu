#include "hip/hip_runtime.h"

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];

	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}

	//YOUR CODE HERE


	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}

	free(A);
	free(B);
	hipFree(d_A);
	hipFree(d_B);

	return 0;
}

__global__
void split(int N, int* input, int* left, int* right){
  int index = threadIdx.x;
  int stride = blockDim.x;

  __shared__ int leftcount;
  __shared__ int rightcount;

  int splitter = input[0];

  for (int i = index+1; i < N-1; i+= stride){
    if(input[i] > splitter){
      right[atomicAdd(&rightcount,1)] = input[i];
    }else{
      left[atomicAdd(&leftcount,1)] = input[i];
    }
  }
  __syncthreads();
  if(index == 0){
    left[atomicAdd(&leftcount,1)] = input[0];
  }

}
 
