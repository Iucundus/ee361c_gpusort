#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"

__global__ void mergesort(int* src, int* dest, int sliceWidth, int size, int slices);
__device__ void merge(int* src, int* dest, int start, int mid, int end);
void swap(int * &a, int * &b);

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];
	
	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}

	int* B = (int *)malloc(size*sizeof(int));

	int threadsPerBlock = 512;
	int blocksPerGrid =((size) + threadsPerBlock - 1) / threadsPerBlock;

	if(size > 1000){
		blocksPerGrid = ((1000) + threadsPerBlock - 1) / threadsPerBlock;
	}

	int *d_A;
	int *d_B;
	hipMalloc((void **) &d_A, size*sizeof(int));
	hipMalloc((void **) &d_B, size*sizeof(int));

	hipMemcpy(d_A,A, size*sizeof(int), hipMemcpyHostToDevice);

	printf("threads: %u\n",blocksPerGrid*threadsPerBlock);

	for(int sliceWidth = 2; sliceWidth < (size*2); sliceWidth = sliceWidth*2){
		int slices = size / ((blocksPerGrid*threadsPerBlock) * sliceWidth) + 1;
		mergesort<<<blocksPerGrid,threadsPerBlock>>>(d_A,d_B,sliceWidth, size, slices);
		swap(d_A,d_B);
	}

	hipMemcpy(A, d_A, size*sizeof(int), hipMemcpyDeviceToHost);

	
	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}
	
	free(A);
	free(B);
	hipFree(d_A);
	hipFree(d_B);
	
	return 0;
}

/*
 * Sets start, midpoint, and end of array for use in merge()
 */
__global__ void mergesort(int* src, int* dest, int sliceWidth, int size, int slices){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	int start = idx*sliceWidth*slices;
	int mid;
	int end;


	for (int slice = 0; slice < slices; slice++) { 
        if (start >= size)
            break;
 
        mid = min(start + (sliceWidth/2), size);
        end = min(start + sliceWidth, size);
        merge(src,dest,start,mid,end);
        start += sliceWidth;
    }

	

}

/*
 * Merges two sorted halves of array into one sorted array
 */
__device__ void merge(int* src, int* dest, int start, int mid, int end){
	int frontIDX = start;
	int backIDX = mid;

	for(int mergeIDX = start; mergeIDX < end; mergeIDX++){
		if(((frontIDX < mid) && (src[frontIDX] < src[backIDX])) || ((frontIDX < mid) && (backIDX >= end))){ //frontIDX element is lower or back array has already been traversed
			dest[mergeIDX] = src[frontIDX];
			frontIDX++;
		}
		else{
			dest[mergeIDX] = src[backIDX];
			backIDX++;
		}
	}

}

/*
 * Swaps array pointers
 */

void swap(int * &a, int * &b){
	int *temp = a;
	a = b;
	b = temp;
}