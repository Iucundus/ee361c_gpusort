#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"

__global__ void mergesort(int* src, int* dest, int sliceWidth, int size);
__device__ void merge(int* src, int* dest, int start, int mid, int end);
<<<<<<< HEAD
=======
//int min(int x, int y);
>>>>>>> 173dc0baa2b18c03682c4057926308ebb32a188f
void swap(int * &a, int * &b);

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];
	
	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}
<<<<<<< HEAD
=======
	
	//YOUR CODE HERE
>>>>>>> 173dc0baa2b18c03682c4057926308ebb32a188f

	int* B = (int *)malloc(size*sizeof(int));

	int threadsPerBlock = 512;
	int blocksPerGrid =((size) + threadsPerBlock - 1) / threadsPerBlock;

	int *d_A;
	int *d_B;
	hipMalloc((void **) &d_A, size*sizeof(int));
	hipMalloc((void **) &d_B, size*sizeof(int));

	hipMemcpy(d_A,A, size*sizeof(int), hipMemcpyHostToDevice);

	for(int sliceWidth = 2; sliceWidth < (size*2); sliceWidth = sliceWidth*2){
		mergesort<<<blocksPerGrid,threadsPerBlock>>>(d_A,d_B,sliceWidth, size);
		swap(d_A,d_B);
	}

	hipMemcpy(A, d_A, size*sizeof(int), hipMemcpyDeviceToHost);

	
	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}
	
	free(A);
	free(B);
	hipFree(d_A);
	hipFree(d_B);
	
	return 0;
}

<<<<<<< HEAD

/*
 * Sets start, midpoint, and end of array for use in merge()
 */

=======
>>>>>>> 173dc0baa2b18c03682c4057926308ebb32a188f
__global__ void mergesort(int* src, int* dest, int sliceWidth, int size){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	int start = idx*sliceWidth;
	int mid = min(start + (sliceWidth/2), size);
	int end = min(start + sliceWidth, size);

	if(start < size){
		//printf("merging %u to %u\n", start, end-1); //debug
		merge(src,dest,start,mid,end);
	}

	

}

<<<<<<< HEAD
/*
 * Merges two sorted halves of array into one sorted array
 */

=======
>>>>>>> 173dc0baa2b18c03682c4057926308ebb32a188f
__device__ void merge(int* src, int* dest, int start, int mid, int end){
	int frontIDX = start;
	int backIDX = mid;

	for(int mergeIDX = start; mergeIDX < end; mergeIDX++){
		if(((frontIDX < mid) && (src[frontIDX] < src[backIDX])) || ((frontIDX < mid) && (backIDX >= end))){ //frontIDX element is lower or back array has already been traversed
			dest[mergeIDX] = src[frontIDX];
			frontIDX++;
		}
		else{
			dest[mergeIDX] = src[backIDX];
			backIDX++;
		}
	}

}

<<<<<<< HEAD
/*
 * Swaps array pointers
 */
=======
// int min(int x, int y){
// 	if(x<y)
// 		return x;
// 	else
// 		return y;
// }
>>>>>>> 173dc0baa2b18c03682c4057926308ebb32a188f

void swap(int * &a, int * &b){
	int *temp = a;
	a = b;
	b = temp;
<<<<<<< HEAD
}
=======
}
>>>>>>> 173dc0baa2b18c03682c4057926308ebb32a188f
