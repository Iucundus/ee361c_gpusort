#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];
	
	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}
	
	//YOUR CODE HERE
	
	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}
	
	free(A);
	
	return 0;
}