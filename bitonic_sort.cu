#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "file_helper.h"

#define THREADS_PER_BLOCK 1024

__global__ void bitonic_sort(int* arrayIn, int* arrayOut, int arrayLen, int chunkSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < arrayLen) {
		int myValue = arrayIn[idx];
		int chunkStart = (idx / chunkSize) * chunkSize;
		int chunkMid = chunkStart + (chunkSize / 2);
		int partnerIndex = chunkSize - (idx - chunkStart) - 1 + chunkStart;
		if (partnerIndex < arrayLen) {
			int partnerValue = arrayIn[partnerIndex];
			int min = (myValue <= partnerValue) ? myValue:partnerValue;
			int max = (myValue > partnerValue) ? myValue:partnerValue;
			myValue = (idx < chunkMid) ? min:max;
		}
		arrayOut[idx] = myValue;
	}
}

__global__ void bitonic_merge(int* arrayIn, int* arrayOut, int arrayLen, int chunkSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < arrayLen) {
		int myValue = arrayIn[idx];
		int chunkStart = (idx / chunkSize) * chunkSize;
		int chunkMid = chunkStart + (chunkSize / 2);
		int partnerIndex = (((idx - chunkStart) + (chunkSize / 2)) % chunkSize) + chunkStart;
		if (partnerIndex < arrayLen) {
			int partnerValue = arrayIn[partnerIndex];
			int min = (myValue <= partnerValue) ? myValue:partnerValue;
			int max = (myValue > partnerValue) ? myValue:partnerValue;
			myValue = (idx < chunkMid) ? min:max;
		}
		arrayOut[idx] = myValue;
	}
}

int main(int argc, char* argv[]) {
	//Set default file name
	const char* inputFile = "inp.txt";
	const char* outputFile = "out.txt";
	//Overwrite file names
	if(argc >= 2)
		inputFile = argv[1];
	if(argc >= 3)
		outputFile = argv[2];
	
	//Read in input array
	int* A;
	int size = getArray(inputFile, &A);
	if(size < 0){
		printf("Error getting array from %s\n", inputFile);
		return -1;
	}
	
	//YOUR CODE HERE
	int *d_arrayIn, *d_arrayOut;
	hipMalloc(&d_arrayIn, size * sizeof(int));
	hipMalloc(&d_arrayOut, size * sizeof(int));
	hipMemcpy(d_arrayIn, A, size * sizeof(int), hipMemcpyHostToDevice);

	for(int sortSize = 2; (sortSize / 2) < size; sortSize = sortSize * 2) {
		//Sort
		bitonic_sort<<<(size + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_arrayIn, d_arrayOut, size, sortSize);
		hipDeviceSynchronize();
			
		int *d_temp = d_arrayOut;
		d_arrayOut = d_arrayIn;
		d_arrayIn = d_temp;
		
		
		//Merge
		for(int mergeSize = sortSize / 2; mergeSize > 1; mergeSize = mergeSize / 2){
			bitonic_merge<<<(size + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_arrayIn, d_arrayOut, size, mergeSize);
			hipDeviceSynchronize();
			
			int *d_temp = d_arrayOut;
			d_arrayOut = d_arrayIn;
			d_arrayIn = d_temp;
		}
	}
	
	hipMemcpy(A, d_arrayIn, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_arrayIn);
	hipFree(d_arrayOut);
	
	//Output array to file
	if(saveArray(outputFile, A, size) < 0) {
		printf("Error writing sorted array to %s\n", outputFile);
	}
	
	free(A);
	
	return 0;
}